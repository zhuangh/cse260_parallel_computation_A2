
#include <hip/hip_runtime.h>
void setGrid(int n, dim3 &blockDim, dim3 &gridDim)
{
   // set your block dimensions and grid dimensions here
   gridDim.x = n / blockDim.x;
   gridDim.y = n / blockDim.y;
   if(n % blockDim.x != 0)
   	gridDim.x++;
   if(n % blockDim.y != 0)
    	gridDim.y++;
}
